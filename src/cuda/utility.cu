#include <iostream>

namespace jusha {
  void check_cuda_error(const char *kernelname, const char *file, int line_no, hipStream_t stream )
  {
#ifdef DEBUG
    check_cuda_error_always(kernelname, file, line_no, stream);
#endif
  }

  void check_cuda_error_always(const char *kernelname, const char *file, int line_no, hipStream_t stream )
  {
    hipError_t err;
    if (stream)
    hipStreamSynchronize(stream);
    else
      err = hipDeviceSynchronize();
    
    err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "CUDA error at kernel " << kernelname << " @ file " << file << " line " << line_no << " for reason: "  << 
	hipGetErrorString(err) << std::endl;
      //    printBacktrace(10);
      abort();
  }
  

  }
}
