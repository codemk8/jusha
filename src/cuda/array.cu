#include "hip/hip_runtime.h"
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/fill.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/functional.h>
#include <algorithm>
#include "./for_each.hpp"
#include "./array.h"

namespace jusha {
  namespace cuda {
    // A simple kernel to initialize a batch of (ptr, size) pairs.
    template <typename Batch, typename T>
    __global__ void batch_fill_kernel(int num_arrays, int num_big_arrays, Batch batch)
    {
      int id = blockIdx.x;
      
      // small arrays are done by one block each
      if (id < num_arrays) {
        T *ptr = batch.ptrs[id];
        T val = batch.vals[id];
        size_t size = batch.sizes[id];
        for (size_t tid = threadIdx.x; tid < size; tid += blockDim.x) {
          ptr[tid] = val;
        }
      }
      // all block takes part in initializing big array
      for (int big_array = 0; big_array < num_big_arrays; big_array++) {
        T *ptr = batch.big_ptrs[big_array];
        T val = batch.vals2[big_array];
        size_t size = batch.big_sizes[big_array];
        for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
             tid < size; tid += blockDim.x * gridDim.x) {
          ptr[tid] = val;
        }
      }
    }

    template <typename T, int BATCH>
    void batch_fill_wrapper(int num_arrays, int num_big_arrays, const BatchInit<T, BATCH> &init, hipStream_t stream)
    {
      int blocks = num_arrays;
      if (num_big_arrays)
        blocks = std::max(64, blocks);
      if (blocks > 0)
        batch_fill_kernel<BatchInit<T, BATCH>, T> 
          <<<blocks, 1024, 0, stream>>>(num_arrays, num_big_arrays, init);
    }

    template void batch_fill_wrapper(int num_arrays, int num_big_arrays, const BatchInit<float, 4> &init, hipStream_t stream);
    template void batch_fill_wrapper(int num_arrays, int num_big_arrays, const BatchInit<float, 8> &init, hipStream_t stream);
    template void batch_fill_wrapper(int num_arrays, int num_big_arrays, const BatchInit<float, 12> &init, hipStream_t stream);
    template void batch_fill_wrapper(int num_arrays, int num_big_arrays, const BatchInit<float, 16> &init, hipStream_t stream);
    template void batch_fill_wrapper(int num_arrays, int num_big_arrays, const BatchInit<float, 20> &init, hipStream_t stream);

    template class BatchInitializer<float, 4>;
    template class BatchInitializer<float, 8>;
    template class BatchInitializer<float, 12>;
    template class BatchInitializer<float, 16>;
    template class BatchInitializer<float, 20>;

    template void batch_fill_wrapper(int num_arrays, int num_big_arrays, const BatchInit<double, 4> &init, hipStream_t stream);
    template void batch_fill_wrapper(int num_arrays, int num_big_arrays, const BatchInit<double, 8> &init, hipStream_t stream);
    template void batch_fill_wrapper(int num_arrays, int num_big_arrays, const BatchInit<double, 12> &init, hipStream_t stream);
    template void batch_fill_wrapper(int num_arrays, int num_big_arrays, const BatchInit<double, 16> &init, hipStream_t stream);
    template void batch_fill_wrapper(int num_arrays, int num_big_arrays, const BatchInit<double, 20> &init, hipStream_t stream);

    template class BatchInitializer<double, 4>;
    template class BatchInitializer<double, 8>;
    template class BatchInitializer<double, 12>;
    template class BatchInitializer<double, 16>;
    template class BatchInitializer<double, 20>;
  }



  /*********************************************************************************
         Multiply
   *********************************************************************************/
  // Implementation
  template <class T>
  void multiply(const JVector<T> &x0, const JVector<T> &x1, JVector<T> &y)
  {
    //    thrust::transform(x0.gbegin(), x0.gend(), x1.gbegin(), y.gbegin(), [](double v1, double v2)->double { return v1 * v2; });
    thrust::transform(x0.gbegin(), x0.gend(), x1.gbegin(), y.gbegin(), thrust::multiplies<T>());
    check_cuda_error("array multiply", __FILE__, __LINE__);
  }

  // Instantiation
  template
  void multiply(const JVector<double> &x0, const JVector<double> &x1, JVector<double> &y);
  template
  void multiply(const JVector<float> &x0, const JVector<float> &x1, JVector<float> &y);
  template
  void multiply(const JVector<int> &x0, const JVector<int> &x1, JVector<int> &y);
  
  
  /*********************************************************************************
         scale
   *********************************************************************************/
  namespace cuda {
    template <class T>
    void MirroredArray<T>::scale(const T &ratio) {
      thrust::transform(gbegin(), gend(), thrust::constant_iterator<T>(ratio), gbegin(), thrust::multiplies<T>());
      check_cuda_error("array scale", __FILE__, __LINE__);
    }
    
    // Instantiation
    template void MirroredArray<double>::scale(const double &ratio);
    template void MirroredArray<float>::scale(const float &ratio);
    template void MirroredArray<int>::scale(const int &ratio);  
  }


  
  /*********************************************************************************
         setVal
   *********************************************************************************/

  namespace cuda {
#if 0      
    template <class T>
    void MirroredArray<T>::fill(const T &val) {
      if (isGpuArray) {
	thrust::fill(gbegin(), gend(), val);
	check_cuda_error("array fill", __FILE__, __LINE__);
      } else {
	std::fill(getPtr(), getPtr()+size(), val);
      }
    }

#endif
    template <typename T>
    void fill(T *begin, T *end, const T & val) {
      thrust::fill(begin, end, val);      
    }

  
  template <class T>
    class fill_run_nv: public nvstd::function<void(T)> {
    public:
      __device__ void operator()(int gid, thrust::tuple<T*, T> &tuple) const {
        thrust::get<0>(tuple)[gid] = thrust::get<1>(tuple);
      }
    };

    template <typename T>
    void fill(thrust::device_ptr<T> begin, thrust::device_ptr<T> end, const T&val)
    {
#if 0 // thrust call
      thrust::fill(begin, end, val);
#else
      ForEachKernel<StridePolicy, JC_cuda_blocksize, false> kernel(end-begin, "Fill"); 
      kernel.run<fill_run_nv<T>, T*, T>(thrust::raw_pointer_cast(begin), val);
#endif
    }
    
    // // Instantiation
    // template void MirroredArray<double>::fill(const double &ratio);
    // template void MirroredArray<float>::fill(const float &ratio);
    // template void MirroredArray<int>::fill(const int &ratio);
    template void fill(bool *, bool *, const bool &);
    template void fill(double *, double *, const double &);
    template void fill(float *, float *, const float &);
    template void fill(int *, int *, const int &);
    template void fill(long long*, long long*, const long long &);
    template void fill(float2 *, float2 *, const float2 &);
    template void fill(float4 *, float4 *, const float4 &);    

    template void fill(thrust::device_ptr<bool> begin, thrust::device_ptr<bool> end, const bool&val);
    template void fill(thrust::device_ptr<double> begin, thrust::device_ptr<double> end, const double&val);
    template void fill(thrust::device_ptr<float> begin, thrust::device_ptr<float> end, const float&val);
    template void fill(thrust::device_ptr<int> begin, thrust::device_ptr<int> end, const int&val);
    template void fill(thrust::device_ptr<unsigned int> begin, thrust::device_ptr<unsigned int> end, const unsigned int&val);
    template void fill(thrust::device_ptr<float2> begin, thrust::device_ptr<float2> end, const float2&val);
    template void fill(thrust::device_ptr<float4> begin, thrust::device_ptr<float4> end, const float4&val);

  }


  /*********************************************************************************
   *      addConst
   *********************************************************************************/
  // Implementation
  template <class T>
  void addConst(JVector<T> &vec, T val)
  {
    thrust::transform(vec.gbegin(), vec.gend(),  thrust::make_constant_iterator(val),
                      vec.gbegin(), thrust::plus<T>());
  }
  // Instantiation
  template void addConst(JVector<int> &vec, int);
  template void addConst(JVector<double> &vec, double);
  template void addConst(JVector<long long> &vec, long long);  
  template void addConst(JVector<float> &vec, float);
  template void addConst(JVector<long> &vec, long);  



  /*********************************************************************************
         norm
   *********************************************************************************/
  template <typename T>
struct square
  {
    __host__ __device__
    T operator()(const T& x) const { 
      return x * x;
    }
  };
  // Implementation
  template <class T>
    T norm(const JVector<T> &vec)
  {
    if (!vec.size()) return 0.0;
    // prefer GPU implementation
    if (vec.GpuHasLatest())  {
      square<T>        unary_op;
      thrust::plus<T> binary_op;
      return std::sqrt( thrust::transform_reduce(vec.gbegin(), vec.gend(), unary_op, 0.0, binary_op) );
    } else {
      assert(vec.CpuHasLatest());
      T sum(0.0);
      const T* vec_ptr = vec.getReadOnlyPtr();
      for (int i = 0; i != vec.size(); i++)
        sum += vec_ptr[i] * vec_ptr[i];
      return std::sqrt(sum);
    }
  }

  // Instantiation
  template float norm (const JVector<float> &vec);
  template double norm (const JVector<double> &vec);

  /*********************************************************************************
         Next
   *********************************************************************************/
  // Implementation
  // Instantiation

}
