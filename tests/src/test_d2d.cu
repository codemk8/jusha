#include "hip/hip_runtime.h"
#include <sstream>
#include <cstdio>
#include <algorithm>

#include <hip/hip_vector_types.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#include "timer.h"
#include "utility.h"
#include "cuda/cuda_config.h"

// #include "cuda/d2d_copy.h"

// include different kernel implementations
#include "cuda/cuda_types.h"
#include "./cudamemcpy.cu"
#include "./d2d_direct.cu"
#include "./d2d_unroll.cu"
#include "./d2d_prefetch.cu"
#include "cuda/array.h"
#include "cuda/test/util.h"

enum test_d2d_kernel_type
{
  test_d2d_cudamemcpy = 0,
  test_d2d_direct = 1,
  test_d2d_unroll = 2,
  test_d2d_prefetch = 3,
  test_d2d_opt = 5
};

#define NOVERIFY

template <typename T>
void test_d2d_cuda(test_d2d_kernel_type type, const char *case_name, size_t byte_size, int offset = 0)
{
  size_t _size = byte_size / sizeof(T);
  jusha::cuda::MirroredArray<T> d_src_vec(_size);
  jusha::cuda::MirroredArray<T> d_dst_vec(_size);
  //
  d_src_vec.zero();
  d_dst_vec.zero();

  // thrust::device_vector<T> d_src_vec(_size);
  // thrust::device_vector<T>  d_dst_vec(_size);

#ifndef NOVERIFY
  thrust::sequence(d_src_vec.begin(), d_src_vec.end(), T());
  thrust::sequence(d_dst_vec.begin(), d_dst_vec.end(), 20);
#endif
  // #ifndef NOVERIFY
  // {
  //   bool equal = jusha::cuda::is_equal(d_src_vec.data(), d_src_vec.data(), d_dst_vec.data());
  //   jassert(equal);
  // }
  // #endif

  std::stringstream sstm;
  sstm << case_name << "_" << _size * sizeof(T);

  _size -= offset;
  jusha::cuda_event_start(sstm.str().c_str());

  switch (type)
  {
  case test_d2d_cudamemcpy:
    d2d_cudamemcpy<T>(d_dst_vec.getGpuPtr() + offset, d_src_vec.getReadOnlyGpuPtr() + offset, _size);
    break;
  case test_d2d_direct:
    d2d_direct(d_dst_vec.getGpuPtr() + offset, d_src_vec.getReadOnlyGpuPtr() + offset, _size);
    ;
    break;
  // case test_d2d_unroll:
  //   d2d_unroll<T, 2>(thrust::raw_pointer_cast(d_dst_vec.data()), thrust::raw_pointer_cast(d_src_vec.data()), _size);
  //   break;
  case test_d2d_prefetch:
    d2d_prefetch<T>(d_dst_vec.getGpuPtr() + offset, d_src_vec.getReadOnlyGpuPtr() + offset, _size);
    break;
  default:;
  }
  jusha::cuda_event_stop(sstm.str().c_str());

#ifndef NOVERIFY
  bool equal = jusha::cuda::is_equal(d_src_vec.data(), d_src_vec.data(), d_dst_vec.data());
  jassert(equal);
#endif
  // jusha::check_cuda_error("after sequence", __FILE__, __LINE__  // printf("here 3.\n");
  // thrust::generate(d_dst_vec.begin(), d_dst_vec.end(), rand);
  // printf("here 4.\n");
  // std::stringstream sstm;
  // sstm << case_name << " " << _size;
  // jusha::cuda_event_start(sstm.str().c_str());
  // jusha::check_cuda_error("after sequence", __FILE__, __LINE__);
  // printf("here 4.\n");
}

int main(int argc, char **argv)
{
  size_t test_size = (2 << 26);
  size_t test_max_size = (2 << 27);
  int type(-1);
  if (argc == 3)
  {
    int min_shift, max_shift;
    min_shift = atoi(argv[1]);
    max_shift = atoi(argv[2]);
    test_size = 2 << min_shift;
    test_max_size = 2 << max_shift;
  }

  if (argc >= 4)
  {
    type = atoi(argv[1]);
    int min_shift, max_shift;
    min_shift = atoi(argv[2]);
    max_shift = atoi(argv[3]);
    test_size = 2 << min_shift;
    test_max_size = 2 << max_shift;
  }
  int offset = 0;
  if (argc == 5)
    offset = atoi(argv[4]);

  jusha::cuda::MirroredArray<unsigned int> first_malloc(20);
  first_malloc.randomize();

  //  size_t test_size = (2<<19);
  const int num_runs = 101;
  for (; test_size < test_max_size; test_size <<= 1)
  {
    std::cout << "test size " << test_size << std::endl;
    int runs = num_runs;
    if (type == test_d2d_cudamemcpy || type < 0)
    {
      runs = num_runs;
      while (runs--)
        test_d2d_cuda<char>(test_d2d_cudamemcpy, "CudaMemcpy_1char", test_size, offset);

      runs = num_runs;
      while (runs--)
        test_d2d_cuda<short>(test_d2d_cudamemcpy, "CudaMemcpy_2short", test_size, offset);

      runs = num_runs;
      while (runs--)
        test_d2d_cuda<int>(test_d2d_cudamemcpy, "CudaMemcpy_3int", test_size, offset);

      runs = num_runs;
      while (runs--)
        test_d2d_cuda<double>(test_d2d_cudamemcpy, "CudaMemcpy_4double", test_size, offset);

#ifdef NOVERIFY
      runs = num_runs;
      while (runs--)
        test_d2d_cuda<float3>(test_d2d_cudamemcpy, "CudaMemcpy_5float3", test_size, offset);

      runs = num_runs;
      while (runs--)
        test_d2d_cuda<float4>(test_d2d_cudamemcpy, "CudaMemcpy_6float4", test_size, offset);
#endif
    }

    if (type < 0 || type == test_d2d_direct)
    {
      runs = num_runs;
      while (runs--)
        test_d2d_cuda<char>(test_d2d_direct, "DirectKernel_1char", test_size, offset);

      runs = num_runs;
      while (runs--)
        test_d2d_cuda<short>(test_d2d_direct, "DirectKernel_2short", test_size, offset);

      runs = num_runs;
      while (runs--)
        test_d2d_cuda<int>(test_d2d_direct, "DirectKernel_3int", test_size, offset);

      runs = num_runs;
      while (runs--)
        test_d2d_cuda<double>(test_d2d_direct, "DirectKernel_4double", test_size, offset);

#ifdef NOVERIFY
      runs = num_runs;
      while (runs--)
        test_d2d_cuda<float3>(test_d2d_direct, "DirectKernel_5float3", test_size, offset);

      runs = num_runs;
      while (runs--)
        test_d2d_cuda<float4>(test_d2d_direct, "DirectKernel_6float4", test_size, offset);

#endif
    }

    // runs = num_runs;
    // while (runs--)
    //   test_d2d_cuda<int>(test_d2d_unroll, "UnrollKernel_3int", test_size);

#ifdef NOVERIFY

    // runs = num_runs;
    // while (runs--)
    //   test_d2d_texture_cuda<float4>(test_d2d_unroll, "Unroll2Kernel_6float4", test_size);

    // runs = num_runs;
    // while (runs--)
    //   test_d2d_cuda<double>(test_d2d_unroll, "Unroll2Kernel_4double", test_size);
#endif
    if (type < 0 || type == test_d2d_prefetch)
    {
      runs = num_runs;
      while (runs--)
        test_d2d_cuda<double>(test_d2d_prefetch, "PrefetchKernel_4double", test_size);

      runs = num_runs;
      while (runs--)
        test_d2d_cuda<float4>(test_d2d_prefetch, "PrefetchKernel_6float4", test_size);
    }
  }
  jusha::cuda_event_print(true);
  return 0;
}
