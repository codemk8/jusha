
#include <hip/hip_runtime.h>
template <typename T>
void d2d_cudamemcpy(void *dst, const void *src, size_t _size)
{
  hipMemcpy(dst, src, _size*sizeof(T), hipMemcpyDefault);
}
