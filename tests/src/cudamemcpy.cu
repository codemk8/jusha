
#include <hip/hip_runtime.h>
template <typename T>
void d2d_cudamemcpy(T *dst, const T *src, size_t _size)
{
  hipMemcpy(dst, src, _size*sizeof(T), hipMemcpyDefault);
}
