#include "hip/hip_runtime.h"
#include <catch.hpp>
#include "utility.h"
#include "cuda/for_each.hpp"
#include "cuda/cuda_config.h"
#include "cuda/array.h"

using namespace jusha;

__global__ void kernel(int N)
{
  ForEach<StridePolicy, 128, false> for_each(N, threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);
  
}

__device__ void atomic_run(int gid) {
}

template <class Fn>
class AtomicAdd: public ForEachKernel<StridePolicy, /*Fn,*/ JC_cuda_blocksize, false> 
{
public:
  explicit AtomicAdd(Fn method, int N): ForEachKernel<StridePolicy, /*Fn,*/JC_cuda_blocksize, false>(N /*,method*/){
  }
  
  virtual __device__ void do_1
() {}  

};

TEST_CASE( "ForEach", "[sum]" ) {
  JVector<int> sum(1);
  sum.zero();
  //  ForEachKernel<StridePolicy, 256, false> fe(300);
  //  AtomicAdd kernel(300);
  auto lambda_func = []() {};
  AtomicAdd<decltype(atomic_run)> kernel(atomic_run, 3);

  printf("running atomic add kernel\n");
  kernel.run(2, sum.getGpuPtr(), sum.getReadOnlyPtr());
  kernel.run(sum.getGpuPtr(), 2, sum.getReadOnlyPtr());
  //  kernel.run();
  //  generic_kernel<<<1,1>>>(sum.getGpuPtr());
  //  fe.run(sum.getGpuPtr());
}

TEST_CASE( "ForEach2", "[wrapper]" ) {
  JVector<int> sum(1);
  sum.zero();
  //  ForEachKernel<StridePolicy, 256, false> fe(300);
  //  AtomicAdd kernel(300);
  KernelWrapper kernel;
  kernel.run(3);
  //  kernel.run(sum.getGpuPtr());
  //  kernel.run();
  //  generic_kernel<<<1,1>>>(sum.getGpuPtr());
  //  fe.run(sum.getGpuPtr());
}
