#include "hip/hip_runtime.h"
#include <catch2/catch.hpp>
#include "utility.h"

#include "cuda/cuda_config.h"
#include "cuda/array.h"

using namespace jusha;

template <class Tuple>
static __device__ void global_for_each(int gid, Tuple &tuple) {
  //   printf("first  %d.\n", std::get<0>(tuple));
  //   printf("second %p.\n", std::get<1>(tuple));
  //   printf("third %p.\n", std::get<2>(tuple));

  // printf("gid %d.\n", gid);
}

#include "cuda/for_each.hpp"


__global__ void kernel(int N)
{
  ForEach<StridePolicy, 128, false> for_each(N, threadIdx.x + blockIdx.x * blockDim.x, blockDim.x * gridDim.x);
  
}

//template <class Tuple>
#if 0
class atomic_run_nv: public nvstd::function<void(int)> {
public:
__device__ void operator()(int gid, std::tuple<int, int *> &tuple) const {
  // printf("first  %d.\n", std::get<0>(tuple));
  // printf("second %p.\n", std::get<1>(tuple));
  atomicAdd(std::get<1>(tuple), std::get<0>(tuple));
}
};

__device__ void atomic_run(int gid) {
    printf("gid %d.\n", gid);
}
#endif

template <class T>
class atomic_run_nv: public nvstd::function<void(T)> {
public:
  __device__ void operator()(int gid, thrust::tuple<T*, T> &tuple) const {
    atomicAdd(thrust::get<0>(tuple), thrust::get<1>(tuple));
  }
};




//template <class Fn>
class AtomicAdd: public ForEachKernel<StridePolicy, JC_cuda_blocksize, false> 
{
public:
  explicit AtomicAdd(int N): ForEachKernel<StridePolicy, JC_cuda_blocksize, false>(N, "AtomicAdd"){
  }
};

#if 1
TEST_CASE( "ForEachStride", "[sum]" ) {
  JVector<int> sum(1);
  sum.zero();
  //  ForEachKernel<StridePolicy, 256, false> fe(300);
  //  AtomicAdd kernel(300);
  int n1 = 3;
  int add_per_thread = 2;
  //  AtomicAdd/*<decltype(atomic_run)>*/ kernel(/*atomic_run,*/ n1);
  ForEachKernel<StridePolicy, JC_cuda_blocksize, false> kernel(n1, "AtomicAdd");
//atomic_run_nv nv_run;
//  printf("running atomic add kernel\n");
  kernel.run<atomic_run_nv<int>, int *, int>(sum.getGpuPtr(), add_per_thread);
  int sum_now = sum[0];
  check_cuda_error("atomic", __FILE__, __LINE__);
  REQUIRE(sum_now == n1*add_per_thread);

  kernel.set_N(257);
  kernel.run<atomic_run_nv<int>, int *, int >(sum.getGpuPtr(), 12);
  REQUIRE(sum[0] == (sum_now + 257*12));

  //  kernel.run(sum.getGpuPtr(), 2, sum.getReadOnlyPtr());
  //  kernel.run();
  //  generic_kernel<<<1,1>>>(sum.getGpuPtr());
  //  fe.run(sum.getGpuPtr());
  check_cuda_error("atomic", __FILE__, __LINE__);
}
#endif


#if 0
TEST_CASE( "ForEach2", "[wrapper]" ) {
  JVector<int> sum(1);
  sum.zero();
  //  ForEachKernel<StridePolicy, 256, false> fe(300);
  //  AtomicAdd kernel(300);
  KernelWrapper kernel;
  kernel.run(3);
  //  kernel.run(sum.getGpuPtr());
  //  kernel.run();
  //  generic_kernel<<<1,1>>>(sum.getGpuPtr());
  //  fe.run(sum.getGpuPtr());
}
#endif


