#include "hip/hip_runtime.h"
#include <catch.hpp>
#include "utility.h"

#include "cuda/cuda_config.h"
#include "cuda/array.h"
#include "cuda/ForEachShmKernel.hpp"
#include "cuda/for_each.hpp"
#include "cuda/cuda_intrinsic.hpp"

using namespace jusha;

// template <class T>
// struct ShReduce {
//   T *sh_ptr;
// };

template <class T>
class reduce_run_nv: public nvstd::function<void(T)> {
public:
  __device__ reduce_run_nv() {
    m_reduce = {};
  }
  __device__ void operator()(int gid, thrust::tuple<const T*, T*> &tuple)  {
    m_reduce += (thrust::get<0>(tuple))[gid];
    //    atomicAdd(thrust::get<0>(tuple), thrust::get<1>(tuple));
  }

  __device__ void post_proc(int gid, thrust::tuple<const T*, T*> &tuple)  {
    //    printf("here in post proc\n");
    // if (blockIdx.x == 0 & threadIdx.x < 2)
    //   printf("my reduce %d tid %d.\n", m_reduce, threadIdx.x);
    m_reduce = jusha::cuda::blockReduceSum(m_reduce);
    if (threadIdx.x == 0)
      (thrust::get<1>(tuple))[blockIdx.x] = m_reduce;

    // if (blockIdx.x == 0 & threadIdx.x < 2)
    //   printf("after my reduce %d tid %d.\n", m_reduce, threadIdx.x);

  }
private:
  T m_reduce;
};



TEST_CASE( "ForEachShmReduce", "[sum]" ) {
   int n = 2000;
   JVector<int> sum(n);
   thrust::fill(sum.gbegin(), sum.gend(), 1);
   //  ForEachKernel<StridePolicy, 256, false> fe(300);
   //  AtomicAdd kernel(300);
   //  AtomicAdd/*<decltype(atomic_run)>*/ kernel(/*atomic_run,*/ n1);
   ForEachShmKernel<BlockPolicy, JC_cuda_warpsize, false> kernel(n, "Reduction");
   kernel.set_block_size(1024);
   kernel.set_max_block(1024);
   JVector<int> inter_sum(1024);
   constexpr int shared_bsize = sizeof(int)*1024/32;
   
   kernel.run<reduce_run_nv<int>, int, shared_bsize, const int *, int *>(sum.getReadOnlyGpuPtr(), inter_sum.getGpuPtr());
   //   inter_sum.print("intersum");
   kernel.set_N(kernel.get_num_blocks());
   kernel.run<reduce_run_nv<int>, int, shared_bsize, const int *, int *>(inter_sum.getReadOnlyGpuPtr(), inter_sum.getGpuPtr());
   hipDeviceSynchronize();
   //   inter_sum.print("intersum after");
   check_cuda_error("inter_sum", __FILE__, __LINE__);
   
   REQUIRE(inter_sum.getElementAt(0) == n);
   // int sum_now = sum[0];
   // REQUIRE(sum_now == n1*add_per_thread);
   
   // kernel.set_N(257);
   // kernel.run<atomic_run_nv<int>, int *, int >(sum.getGpuPtr(), 12);
   // REQUIRE(sum[0] == (sum_now + 257*12));

   check_cuda_error("inter_sum", __FILE__, __LINE__);
}


